#include "CudaHelper.cuh"

void checkCudaErrors(hipError_t err)
{
	if (err > 0) {
		std::cout << "Error: " << hipGetErrorString(err);
		exit(err);
	}
}