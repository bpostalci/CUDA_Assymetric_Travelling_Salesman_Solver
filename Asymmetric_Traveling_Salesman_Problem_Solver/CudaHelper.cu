#include "CudaHelper.cuh"

void checkCudaErrors(hipError_t err)
{
	if (err > 0) {
		std::cerr << "Error: " << hipGetErrorString(err);
		exit(err);
	}
}